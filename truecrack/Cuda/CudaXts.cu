/*
 * Copyright (C)  2011  Luca Vaccaro
 * Based on TrueCrypt, freely available at http://www.truecrypt.org/
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */
#if BYTE_ORDER == BIG_ENDIAN
#error The TC_NO_COMPILER_INT64 version of the XTS code is not compatible with big-endian platforms
#endif
#include "Endian.h"

#if BYTE_ORDER == LITTLE_ENDIAN
#	define CUDA_BE16(x) cuda_MirrorBytes16(x)
#	define CUDA_BE32(x) cuda_MirrorBytes32(x)
#	define CUDA_BE64(x) cuda_MirrorBytes64(x)
#else
#	define CUDA_BE16(x) (x)
#	define CUDA_BE32(x) (x)
#	define CUDA_BE64(x) (x)
#endif

__device__ unsigned __int16 cuda_MirrorBytes16 (unsigned __int16 x)
{
	return (x << 8) | (x >> 8);
}


__device__ unsigned __int32 cuda_MirrorBytes32 (unsigned __int32 x)
{
	unsigned __int32 n = (unsigned __int8) x;
	n <<= 8; n |= (unsigned __int8) (x >> 8);
	n <<= 8; n |= (unsigned __int8) (x >> 16);
	return (n << 8) | (unsigned __int8) (x >> 24);
}

__device__ uint16 cuda_GetHeaderField16 (byte *header, int offset)
{
	return CUDA_BE16 (*(uint16 *) (header + offset));
}


__device__ uint32 cuda_GetHeaderField32 (byte *header, int offset)
{
	return CUDA_BE32 (*(uint32 *) (header + offset));
}


/* CRC polynomial 0x04c11db7 */
__constant__ unsigned __int32 cuda_crc_32_tab[]=
{				
	0x00000000, 0x77073096, 0xee0e612c, 0x990951ba, 0x076dc419, 0x706af48f, 0xe963a535, 0x9e6495a3,
	0x0edb8832, 0x79dcb8a4, 0xe0d5e91e, 0x97d2d988, 0x09b64c2b, 0x7eb17cbd, 0xe7b82d07, 0x90bf1d91,
	0x1db71064, 0x6ab020f2, 0xf3b97148, 0x84be41de, 0x1adad47d, 0x6ddde4eb, 0xf4d4b551, 0x83d385c7,
	0x136c9856, 0x646ba8c0, 0xfd62f97a, 0x8a65c9ec, 0x14015c4f, 0x63066cd9, 0xfa0f3d63, 0x8d080df5,
	0x3b6e20c8, 0x4c69105e, 0xd56041e4, 0xa2677172, 0x3c03e4d1, 0x4b04d447, 0xd20d85fd, 0xa50ab56b,
	0x35b5a8fa, 0x42b2986c, 0xdbbbc9d6, 0xacbcf940, 0x32d86ce3, 0x45df5c75, 0xdcd60dcf, 0xabd13d59,
	0x26d930ac, 0x51de003a, 0xc8d75180, 0xbfd06116, 0x21b4f4b5, 0x56b3c423, 0xcfba9599, 0xb8bda50f,
	0x2802b89e, 0x5f058808, 0xc60cd9b2, 0xb10be924, 0x2f6f7c87, 0x58684c11, 0xc1611dab, 0xb6662d3d,
	0x76dc4190, 0x01db7106, 0x98d220bc, 0xefd5102a, 0x71b18589, 0x06b6b51f, 0x9fbfe4a5, 0xe8b8d433,
	0x7807c9a2, 0x0f00f934, 0x9609a88e, 0xe10e9818, 0x7f6a0dbb, 0x086d3d2d, 0x91646c97, 0xe6635c01,
	0x6b6b51f4, 0x1c6c6162, 0x856530d8, 0xf262004e, 0x6c0695ed, 0x1b01a57b, 0x8208f4c1, 0xf50fc457,
	0x65b0d9c6, 0x12b7e950, 0x8bbeb8ea, 0xfcb9887c, 0x62dd1ddf, 0x15da2d49, 0x8cd37cf3, 0xfbd44c65,
	0x4db26158, 0x3ab551ce, 0xa3bc0074, 0xd4bb30e2, 0x4adfa541, 0x3dd895d7, 0xa4d1c46d, 0xd3d6f4fb,
	0x4369e96a, 0x346ed9fc, 0xad678846, 0xda60b8d0, 0x44042d73, 0x33031de5, 0xaa0a4c5f, 0xdd0d7cc9,
	0x5005713c, 0x270241aa, 0xbe0b1010, 0xc90c2086, 0x5768b525, 0x206f85b3, 0xb966d409, 0xce61e49f,
	0x5edef90e, 0x29d9c998, 0xb0d09822, 0xc7d7a8b4, 0x59b33d17, 0x2eb40d81, 0xb7bd5c3b, 0xc0ba6cad,
	0xedb88320, 0x9abfb3b6, 0x03b6e20c, 0x74b1d29a, 0xead54739, 0x9dd277af, 0x04db2615, 0x73dc1683,
	0xe3630b12, 0x94643b84, 0x0d6d6a3e, 0x7a6a5aa8, 0xe40ecf0b, 0x9309ff9d, 0x0a00ae27, 0x7d079eb1,
	0xf00f9344, 0x8708a3d2, 0x1e01f268, 0x6906c2fe, 0xf762575d, 0x806567cb, 0x196c3671, 0x6e6b06e7,
	0xfed41b76, 0x89d32be0, 0x10da7a5a, 0x67dd4acc, 0xf9b9df6f, 0x8ebeeff9, 0x17b7be43, 0x60b08ed5,
	0xd6d6a3e8, 0xa1d1937e, 0x38d8c2c4, 0x4fdff252, 0xd1bb67f1, 0xa6bc5767, 0x3fb506dd, 0x48b2364b,
	0xd80d2bda, 0xaf0a1b4c, 0x36034af6, 0x41047a60, 0xdf60efc3, 0xa867df55, 0x316e8eef, 0x4669be79,
	0xcb61b38c, 0xbc66831a, 0x256fd2a0, 0x5268e236, 0xcc0c7795, 0xbb0b4703, 0x220216b9, 0x5505262f,
	0xc5ba3bbe, 0xb2bd0b28, 0x2bb45a92, 0x5cb36a04, 0xc2d7ffa7, 0xb5d0cf31, 0x2cd99e8b, 0x5bdeae1d,
	0x9b64c2b0, 0xec63f226, 0x756aa39c, 0x026d930a, 0x9c0906a9, 0xeb0e363f, 0x72076785, 0x05005713,
	0x95bf4a82, 0xe2b87a14, 0x7bb12bae, 0x0cb61b38, 0x92d28e9b, 0xe5d5be0d, 0x7cdcefb7, 0x0bdbdf21,
	0x86d3d2d4, 0xf1d4e242, 0x68ddb3f8, 0x1fda836e, 0x81be16cd, 0xf6b9265b, 0x6fb077e1, 0x18b74777,
	0x88085ae6, 0xff0f6a70, 0x66063bca, 0x11010b5c, 0x8f659eff, 0xf862ae69, 0x616bffd3, 0x166ccf45,
	0xa00ae278, 0xd70dd2ee, 0x4e048354, 0x3903b3c2, 0xa7672661, 0xd06016f7, 0x4969474d, 0x3e6e77db,
	0xaed16a4a, 0xd9d65adc, 0x40df0b66, 0x37d83bf0, 0xa9bcae53, 0xdebb9ec5, 0x47b2cf7f, 0x30b5ffe9,
	0xbdbdf21c, 0xcabac28a, 0x53b39330, 0x24b4a3a6, 0xbad03605, 0xcdd70693, 0x54de5729, 0x23d967bf,
	0xb3667a2e, 0xc4614ab8, 0x5d681b02, 0x2a6f2b94, 0xb40bbe37, 0xc30c8ea1, 0x5a05df1b, 0x2d02ef8d
};

__device__ unsigned __int32 cuda_GetCrc32 (unsigned char *data, int length)
{
	unsigned __int32 CRC = 0xffffffff;

	while (length--)
	{
		CRC = (CRC >> 8) ^ cuda_crc_32_tab[ (CRC ^ *data++) & 0xFF ];
	}

	return CRC ^ 0xffffffff;
}

__device__ void cuda_memcpy (unsigned char* to , unsigned char* from, int length){
  int i;
  for (i=0;i<length;i++)
    to[i]=from[i];
}


__device__ void cuda_EncipherBlock(int cipher, void *data, void *ks)
{
	switch (cipher)
	{
	case AES:	
		// In 32-bit kernel mode, due to KeSaveFloatingPointState() overhead, AES instructions can be used only when processing the whole data unit.
		aes_encrypt ((const unsigned char*)data, (unsigned char*)data, (const aes_encrypt_ctx *)ks);
		break;

	default:			TC_THROW_FATAL_EXCEPTION;	// Unknown/wrong ID
	}
}
__device__ void cuda_DecipherBlock(int cipher, void *data, void *ks)
{
	switch (cipher)
	{
#ifndef TC_WINDOWS_BOOT

	case AES:
		aes_decrypt ((const unsigned char*)data, (unsigned char*)data, (const aes_decrypt_ctx *) ((char *) ks + sizeof(aes_decrypt_ctx)));
		break;
#else
	case AES:		aes_decrypt ((unsigned char*)data, (unsigned char*)data, ((const aes_decrypt_ctx *))ks); break;
#endif
	default:		TC_THROW_FATAL_EXCEPTION;	// Unknown/wrong ID
	}
}


// Converts a 64-bit unsigned integer (passed as two 32-bit integers for compatibility with non-64-bit
// environments/platforms) into a little-endian 16-byte array.
__device__ static void cuda_Uint64ToLE16ByteArray (unsigned __int8 *byteBuf, unsigned __int32 highInt32, unsigned __int32 lowInt32)
{
	unsigned __int32 *bufPtr32 = (unsigned __int32 *) byteBuf;

	*bufPtr32++ = lowInt32;
	*bufPtr32++ = highInt32;

	// We're converting a 64-bit number into a little-endian 16-byte array so we can zero the last 8 bytes
	*bufPtr32++ = 0;
	*bufPtr32 = 0;
}
    


// Encrypts or decrypts all blocks in the buffer in XTS mode. For descriptions of the input parameters,
// see the 64-bit version of EncryptBufferXTS().
__device__ static void cuda_EncryptDecryptBufferXTS32 (const unsigned __int8 *buffer,
        TC_LARGEST_COMPILER_UINT length,
        const UINT64_STRUCT *startDataUnitNo,
        unsigned int startBlock,
        unsigned __int8 *ks,
        unsigned __int8 *ks2,
        int cipher,
        BOOL decryption)
{

  __align__(8) unsigned __int8 byteBufUnitNo [BYTES_PER_XTS_BLOCK];
  __align__(8) unsigned __int8 whiteningValue [BYTES_PER_XTS_BLOCK];
  __align__(8) unsigned __int8 finalCarry;
  unsigned __int32 *whiteningValuePtr32;
  unsigned __int32 *finalDwordWhiteningValuePtr; 
  unsigned __int32 *bufPtr32;  

  TC_LARGEST_COMPILER_UINT blockCount;
  UINT64_STRUCT dataUnitNo;
  unsigned int block;
  unsigned int endBlock;


	bufPtr32 = (unsigned __int32 *) buffer;
	whiteningValuePtr32 = (unsigned __int32 *) whiteningValue;
	finalDwordWhiteningValuePtr = whiteningValuePtr32 + sizeof (whiteningValue) / sizeof (*whiteningValuePtr32) - 1;


	// Store the 64-bit data unit number in a way compatible with non-64-bit environments/platforms
	dataUnitNo.HighPart = startDataUnitNo->HighPart;
	dataUnitNo.LowPart = startDataUnitNo->LowPart;

	blockCount = length / BYTES_PER_XTS_BLOCK;

	// Convert the 64-bit data unit number into a little-endian 16-byte array. 
	// (Passed as two 32-bit integers for compatibility with non-64-bit environments/platforms.)
	cuda_Uint64ToLE16ByteArray (byteBufUnitNo, dataUnitNo.HighPart, dataUnitNo.LowPart);

	// Generate whitening values for all blocks in the buffer
	while (blockCount > 0)
	{

		
		if (blockCount < BLOCKS_PER_XTS_DATA_UNIT)
			endBlock = startBlock + (unsigned int) blockCount;
		else
			endBlock = BLOCKS_PER_XTS_DATA_UNIT;
		

		// Encrypt the data unit number using the secondary key (in order to generate the first 
		// whitening value for this data unit)
		cuda_Uint64ToLE16ByteArray (byteBufUnitNo, dataUnitNo.HighPart, dataUnitNo.LowPart);
		memcpy (whiteningValue, byteBufUnitNo, BYTES_PER_XTS_BLOCK);
		cuda_EncipherBlock (cipher, whiteningValue, ks2);

		// Generate (and apply) subsequent whitening values for blocks in this data unit and
		// encrypt/decrypt all relevant blocks in this data unit
		for (block = 0; block < endBlock; block++)
		{
			if (block >= startBlock)
			{
				whiteningValuePtr32 = (unsigned __int32 *) whiteningValue;

				// Whitening
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32 ^= *whiteningValuePtr32;

				bufPtr32 -= BYTES_PER_XTS_BLOCK / sizeof (*bufPtr32) - 1;

				// Actual encryption/decryption
				if (decryption)
					cuda_DecipherBlock (cipher, bufPtr32, ks);
				else
					cuda_EncipherBlock (cipher, bufPtr32, ks);

				whiteningValuePtr32 = (unsigned __int32 *) whiteningValue;

				// Whitening
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32++;
				*bufPtr32++ ^= *whiteningValuePtr32;
			}

			// Derive the next whitening value

			finalCarry = 0;

			for (whiteningValuePtr32 = finalDwordWhiteningValuePtr;
				whiteningValuePtr32 >= (unsigned __int32 *) whiteningValue;
				whiteningValuePtr32--)
			{
				if (*whiteningValuePtr32 & 0x80000000)	// If the following shift results in a carry
				{
					if (whiteningValuePtr32 != finalDwordWhiteningValuePtr)	// If not processing the highest double word
					{
						// A regular carry
						*(whiteningValuePtr32 + 1) |= 1;
					}
					else 
					{
						// The highest byte shift will result in a carry
						finalCarry = 135;
					}
				}

				*whiteningValuePtr32 <<= 1;
			}

			whiteningValue[0] ^= finalCarry;
		}

		blockCount -= endBlock - startBlock;
		startBlock = 0;

		// Increase the data unit number by one
		if (!++dataUnitNo.LowPart)
		{
			dataUnitNo.HighPart++;
		}

		// Convert the 64-bit data unit number into a little-endian 16-byte array. 
		cuda_Uint64ToLE16ByteArray (byteBufUnitNo, dataUnitNo.HighPart, dataUnitNo.LowPart);
	}

	FAST_ERASE64 (whiteningValue, sizeof (whiteningValue));
}


// For descriptions of the input parameters, see the 64-bit version of EncryptBufferXTS().
__device__ void cuda_DecryptBufferXTS (unsigned __int8 *buffer,
					   TC_LARGEST_COMPILER_UINT length,
					   const UINT64_STRUCT *startDataUnitNo,
					   unsigned int startCipherBlockNo,
					   unsigned __int8 *ks,
					   unsigned __int8 *ks2,
					   int cipher)
{
	// Decrypt all ciphertext blocks in the buffer
	cuda_EncryptDecryptBufferXTS32 (buffer, length, startDataUnitNo, startCipherBlockNo, ks, ks2, cipher, TRUE);
}

__device__ void cuda_DecryptBuffer (unsigned __int8 *buf, TC_LARGEST_COMPILER_UINT len, PCRYPTO_INFO cryptoInfo)
{
	//unsigned __int8 *ks = cryptoInfo->ks;  //+ EAGetKeyScheduleSize (cryptoInfo->ea);
	//unsigned __int8 *ks2 = cryptoInfo->ks2;// + EAGetKeyScheduleSize (cryptoInfo->ea);
	UINT64_STRUCT dataUnitNo;
	//int cipher;

	// When encrypting/decrypting a buffer (typically a volume header) the sequential number
	// of the first XTS data unit in the buffer is always 0 and the start of the buffer is
	// always assumed to be aligned with the start of the data unit 0.
	dataUnitNo.LowPart = 0;
	dataUnitNo.HighPart = 0;

//	for (cipher = EAGetLastCipher (cryptoInfo->ea);
//		cipher != 0;
//		cipher = EAGetPreviousCipher (cryptoInfo->ea, cipher))
//	{
//		ks -= CipherGetKeyScheduleSize (cipher);
//		ks2 -= CipherGetKeyScheduleSize (cipher);
		cuda_DecryptBufferXTS (buf, len, &dataUnitNo, 0, cryptoInfo->ks, cryptoInfo->ks2, AES);
//	}
}

/* Return values: 0 = success, ERR_CIPHER_INIT_FAILURE (fatal), ERR_CIPHER_INIT_WEAK_KEY (non-fatal) */
__device__ int cuda_CipherInit (int cipher, unsigned char *key, unsigned __int8 *ks)
{
    int retVal = ERR_SUCCESS;

    if (aes_encrypt_key256 (key, (aes_encrypt_ctx *) ks) != EXIT_SUCCESS)
        return ERR_CIPHER_INIT_FAILURE;

    if (aes_decrypt_key256 (key, (aes_decrypt_ctx *) (ks + sizeof(aes_encrypt_ctx))) != EXIT_SUCCESS)
        return ERR_CIPHER_INIT_FAILURE;

    return retVal;
}

// Return values: 0 = success, ERR_CIPHER_INIT_FAILURE (fatal), ERR_CIPHER_INIT_WEAK_KEY (non-fatal)
__device__ int cuda_EAInit (int ea, unsigned char *key, unsigned __int8 *ks)
{
    int c, retVal = ERR_SUCCESS;

    if (ea == 0)
        return ERR_CIPHER_INIT_FAILURE;
    c=AES;
    //for (c = EAGetFirstCipher (ea); c != 0; c = EAGetNextCipher (ea, c))
    //{
    switch (cuda_CipherInit (c, key, ks))
    {
    case ERR_CIPHER_INIT_FAILURE:
        return ERR_CIPHER_INIT_FAILURE;

    case ERR_CIPHER_INIT_WEAK_KEY:
        retVal = ERR_CIPHER_INIT_WEAK_KEY;              // Non-fatal error
        break;
    }

    //key += CipherGetKeySize (c);
    //ks += CipherGetKeyScheduleSize (c);
    //}
    return retVal;
}


__device__ BOOL cuda_EAInitMode (PCRYPTO_INFO ci)
{
	switch (ci->mode)
	{
	case XTS:
		// Secondary key schedule
		if (cuda_EAInit (ci->ea, ci->km2, ci->ks2) != ERR_SUCCESS)
			return FALSE;

		/* Note: XTS mode could potentially be initialized with a weak key causing all blocks in one data unit
		on the volume to be tweaked with zero tweaks (i.e. 512 bytes of the volume would be encrypted in ECB
		mode). However, to create a TrueCrypt volume with such a weak key, each human being on Earth would have
		to create approximately 11,378,125,361,078,862 (about eleven quadrillion) TrueCrypt volumes (provided 
		that the size of each of the volumes is 1024 terabytes). */
		break;
	default:		
		// Unknown/wrong ID
		TC_THROW_FATAL_EXCEPTION;
	}
	return TRUE;
}




__device__ int cuda_Xts(unsigned char *encryptedHeader, unsigned char *headerKey, unsigned char *header) {

    PCRYPTO_INFO cryptoInfo;
    CRYPTO_INFO cryptoInfo_struct;

    uint16 headerVersion;
    int status = ERR_PARAMETER_INCORRECT;
    int primaryKeyOffset=0;

    //int pkcs5PrfCount = LAST_PRF_ID - FIRST_PRF_ID + 1;

    cryptoInfo=&cryptoInfo_struct;
    memset (cryptoInfo, 0, sizeof (CRYPTO_INFO));
    if (cryptoInfo == NULL)
        return ERR_OUT_OF_MEMORY;


    // Support only XTS
    cryptoInfo->mode= XTS ;
    cryptoInfo->ea=AES;

    status = cuda_EAInit (cryptoInfo->ea, headerKey + primaryKeyOffset, cryptoInfo->ks);
    if (status == ERR_CIPHER_INIT_FAILURE)
        return ERR_CIPHER_INIT;
    // Init objects related to the mode of operation

    // Copy the secondary key (if cascade, multiple concatenated)
    //memcpy (cryptoInfo->km2, headerKey + EAGetKeySize (cryptoInfo->ea), EAGetKeySize (cryptoInfo->ea));
    cuda_memcpy (cryptoInfo->km2, headerKey + 32, 32);
    // Secondary key schedule
    if (!cuda_EAInitMode (cryptoInfo)) {
        return ERR_MODE_INIT;
    }
 
    // Copy the header for decryption
    cuda_memcpy (header, encryptedHeader, 512*sizeof(unsigned char));

    // Try to decrypt header
    cuda_DecryptBuffer (header + HEADER_ENCRYPTED_DATA_OFFSET, HEADER_ENCRYPTED_DATA_SIZE, cryptoInfo);
   
    
            // Magic 'TRUE'
            if (cuda_GetHeaderField32 (header, TC_HEADER_OFFSET_MAGIC) != 0x54525545)
                return ERR_MAGIC_TRUE;

            // Header version
            headerVersion = cuda_GetHeaderField16 (header, TC_HEADER_OFFSET_VERSION);
            if (headerVersion > VOLUME_HEADER_VERSION) {
                return ERR_VERSION_REQUIRED;
            }

            // Check CRC of the header fields
            if (headerVersion >= 4
                    && cuda_GetHeaderField32 (header, TC_HEADER_OFFSET_HEADER_CRC) != cuda_GetCrc32 (header + TC_HEADER_OFFSET_MAGIC, TC_HEADER_OFFSET_HEADER_CRC - TC_HEADER_OFFSET_MAGIC))
                //printf("Unsuccessful\n");
                return ERR_CRC_HEADER_FIELDS;
            // Required program version
            //cryptoInfo->RequiredProgramVersion = GetHeaderField16 (header, TC_HEADER_OFFSET_REQUIRED_VERSION);
            //cryptoInfo->LegacyVolume = cryptoInfo->RequiredProgramVersion < 0x600;

            // Check CRC of the key set
            if (cuda_GetHeaderField32 (header, TC_HEADER_OFFSET_KEY_AREA_CRC) != cuda_GetCrc32 (header + HEADER_MASTER_KEYDATA_OFFSET, MASTER_KEYDATA_SIZE))
                return ERR_CRC_KEY_SET;
 
    return SUCCESS;
}



