#include "hip/hip_runtime.h"
/*
 * Copyright (C)  2011  Luca Vaccaro
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "Tcdefs.h"
#include "Volumes.h"
#include <stdio.h>
#include <memory.h>
#include "Crypto.h"
#include "CudaCore.cuh"
#include "CudaPkcs5.cuh"
#include "CudaXts.cuh"



/* The max number of block grid; number of max parallel gpu blocks. */
int blockGridSizeMax;

/* The number of the current block grid; number of current parallel gpu blocks. */
int blockGridSizeCurrent;

/* Pointer of structures to pass to Cuda Kernel. */
unsigned char *dev_salt, *dev_blockPwd, *dev_header;
int *dev_blockPwd_init, *dev_blockPwd_length;
short int *dev_result;


int getMultiprocessorCount (void){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.multiProcessorCount;
}

__global__ void cuda_Kernel ( unsigned char *salt, unsigned char *headerEncrypted, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result) {

    int numData=blockIdx.x;
    int numBlock=threadIdx.x;

    // Array of unsigned char in the shared memory
    __shared__ __align__(8) unsigned char headerkey[192];
    __shared__ __align__(8) unsigned char headerDecrypted[512];
    
    
    // Calculate the hash header key
    cuda_Pbkdf2 (salt, blockPwd, blockPwd_init, blockPwd_length, headerkey, numData, numBlock);

    // Synchronize all threads in the block
    __syncthreads();
    
    // Decrypt the header and compare the key
    if (numBlock==0) {
        int value;
        value=cuda_Xts (headerEncrypted, headerkey,headerDecrypted);

        if (value==SUCCESS)
            result[numData]=MATCH;
        else
            result[numData]=NOMATCH;
    }
    //__syncthreads();
}
void cuda_Core ( short int *result) {


    hipMalloc ( &dev_result, blockGridSizeCurrent * sizeof(short int)) ;
    hipMemcpy( dev_result, result, blockGridSizeCurrent * sizeof(short int) , hipMemcpyHostToDevice) ;

    cuda_Kernel<<<blockGridSizeCurrent,10>>>(dev_salt, dev_header, dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_result);
    //hipDeviceSynchronize();

    hipMemcpy(result, dev_result,blockGridSizeCurrent * sizeof(short int) , hipMemcpyDeviceToHost) ;
}


void cuda_Init (int block_maxsize, unsigned char *salt, unsigned char *header) {
    blockGridSizeMax=block_maxsize;

    hipMalloc ( (void **)&dev_blockPwd, blockGridSizeMax * PASSWORD_MAXSIZE * sizeof(unsigned char)) ;
    hipMalloc ( (void **)&dev_blockPwd_init, blockGridSizeMax * sizeof(int)) ;
    hipMalloc ( (void **)&dev_blockPwd_length, blockGridSizeMax * sizeof(int)) ;
    hipMalloc ( (void **)&dev_salt, SALT_LENGTH * sizeof(unsigned char)) ;
    hipMalloc ( (void **)&dev_header, TC_VOLUME_HEADER_EFFECTIVE_SIZE * sizeof(unsigned char)) ;
    hipMalloc ( (void **)&dev_result, blockGridSizeMax * sizeof(short int)) ;

    hipMemcpy(dev_salt, salt, SALT_LENGTH * sizeof(unsigned char) , hipMemcpyHostToDevice) ;
    hipMemcpy(dev_header, header, TC_VOLUME_HEADER_EFFECTIVE_SIZE * sizeof(unsigned char) , hipMemcpyHostToDevice) ;

}

void cuda_Set (	int block_currentsize, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result) {

    blockGridSizeCurrent=block_currentsize;
    int lengthpwd=0,i;
    for (i=0;i<blockGridSizeCurrent;i++) {
        lengthpwd+=blockPwd_length[i];
        result[i]=NODEFINED;
    }

    hipMemcpy(dev_blockPwd, blockPwd, lengthpwd * sizeof(unsigned char) , hipMemcpyHostToDevice) ;
    hipMemcpy(dev_blockPwd_init, blockPwd_init, blockGridSizeCurrent * sizeof(int) , hipMemcpyHostToDevice) ;
    hipMemcpy(dev_blockPwd_length, blockPwd_length, blockGridSizeCurrent * sizeof(int) , hipMemcpyHostToDevice) ;
    hipMemcpy(dev_result, result, blockGridSizeCurrent * sizeof(short int) , hipMemcpyHostToDevice) ;
}


void cuda_Free(void) {
    hipFree(dev_salt);
    hipFree(dev_blockPwd);
    hipFree(dev_blockPwd_init);
    hipFree(dev_blockPwd_length);
    hipFree(dev_result);
    hipFree(dev_header);
}
