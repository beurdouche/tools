/*
 * Copyright (C)  2011  Luca Vaccaro
 * Based on TrueCrypt, freely available at http://www.truecrypt.org/
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */
#include "CudaPkcs5.cuh"



__device__ void cuda_hmac_ripemd160 (unsigned char *key, int keylen, unsigned char *input, int len, unsigned char *digest, SupportPkcs5 *sup)
{
  
    

    int i;
    // If the key is longer than the hash algorithm block size,
    //	   let key = ripemd160(key), as per HMAC specifications. 
    if (keylen > RIPEMD160_BLOCKSIZE) 
	{
	  //RMD160Init(&tctx);
        //RMD160Update(&tctx, (const unsigned char *) key, keylen);
        //RMD160Final(tk, &tctx);
	cuda_RMD160(&sup->ctctx,(unsigned char *) key, keylen,(unsigned char *)NULL,0,sup->ctk);
    
        key = (unsigned char *) sup->ctk;
        keylen = RIPEMD160_DIGESTSIZE;

	//burn (&ctctx, sizeof(ctctx));	// Prevent leaks
    }

	/*

	RMD160(K XOR opad, RMD160(K XOR ipad, text))

	where K is an n byte key
	ipad is the byte 0x36 repeated RIPEMD160_BLOCKSIZE times
	opad is the byte 0x5c repeated RIPEMD160_BLOCKSIZE times
	and text is the data being protected */


    /* start out by storing key in pads */
    // XOR key with ipad and opad values 

    for (i=0; i<sizeof(sup->cpad); i++) 
	sup->cpad[i]=0x36;
    for (i=0; i<keylen; i++)
        sup->cpad[i] ^= key[i];
   
    cuda_RMD160(&sup->ccontext,sup->cpad,RIPEMD160_BLOCKSIZE,(const unsigned char *) input, len, (unsigned char *) digest);
   
    for (i=0; i<sizeof(sup->cpad); i++) 
	sup->cpad[i]=0x5c; 
    for (i=0; i<keylen; i++) 
	sup->cpad[i] ^= key[i];
    cuda_RMD160(&sup->ccontext,sup->cpad,RIPEMD160_BLOCKSIZE,(const unsigned char *) digest, RIPEMD160_DIGESTSIZE, (unsigned char *) digest);
   

    // perform inner RIPEMD-160

    //RMD160Init(&context);           // init context for 1st pass
    //RMD160Update(&context, k_ipad, RIPEMD160_BLOCKSIZE);  // start with inner pad
    //RMD160Update(&context, (const unsigned char *) input, len); // then text of datagram 
    //RMD160Final((unsigned char *) digest, &context);         // finish up 1st pass 
    //cuda_RMD160(&ccontext,ck_ipad,RIPEMD160_BLOCKSIZE,(const unsigned char *) input, len, (unsigned char *) digest);
   
    // perform outer RIPEMD-160 
    //RMD160Init(&context);           // init context for 2nd pass 
    //RMD160Update(&context, k_opad, RIPEMD160_BLOCKSIZE);  // start with outer pad 
    // results of 1st hash 
    //RMD160Update(&context, (const unsigned char *) digest, RIPEMD160_DIGESTSIZE);
    //RMD160Final((unsigned char *) digest, &context);         // finish up 2nd pass 
    //cuda_RMD160(&ccontext,ck_opad,RIPEMD160_BLOCKSIZE,(const unsigned char *) digest, RIPEMD160_DIGESTSIZE, (unsigned char *) digest);
   
	// Prevent possible leaks. 
	//burn (ck_ipad, sizeof(ck_ipad));
	//burn (ck_opad, sizeof(ck_opad));
	//burn (ctk, sizeof(ctk));
	//burn (&ccontext, sizeof(ccontext));
}


__device__ void cuda_Pbkdf2 ( unsigned char *salt, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, unsigned char *headerkey, int numData, int numBlock) {
	SupportPkcs5 support;
	SupportPkcs5 *sup;
	sup = &support;
	
	//INCLUDE: void derive_u_ripemd160 (char *pwd, int pwd_len, char *salt, int salt_len, int iterations, char *u, int b)
	unsigned char *pwd;
	int pwd_len;
	int c, i;
		
	pwd=blockPwd+blockPwd_init[numData];
	pwd_len = blockPwd_length[numData];
	
	
	int b=numBlock;
	unsigned char *u=headerkey+RIPEMD160_DIGESTSIZE*b;

	// iteration 1 
	memset (sup->ccounter, 0, 4);
	sup->ccounter[3] = (char) b+1;
	memcpy (sup->cinit, salt, SALT_LENGTH);	// salt 
	memcpy (&sup->cinit[SALT_LENGTH],sup->ccounter, 4);	// big-endian block number 
	
	cuda_hmac_ripemd160 (pwd, pwd_len, sup->cinit, SALT_LENGTH + 4, sup->cj, sup);
	memcpy (u, sup->cj, RIPEMD160_DIGESTSIZE);
	
	//remaining iterations 
	for (c = 1; c < ITERATIONS; c++)
	{
		cuda_hmac_ripemd160 (pwd, pwd_len, sup->cj, RIPEMD160_DIGESTSIZE, sup->ck,sup);
		for (i = 0; i < RIPEMD160_DIGESTSIZE; i++)
		{
			u[i] ^= sup->ck[i];
			sup->cj[i] = sup->ck[i];
		}
	}
  
}
